/*
* PSCF++ Package 
*
* Copyright 2010 - 2017, The Regents of the University of Minnesota
* Distributed under the terms of the GNU General Public License.
*/

#include "FFT.tpp"

namespace Pscf {
namespace Pspg {

   using namespace Util;

   // Forward transform, explicit specializations.
   //use local mesh dimensions later
   template<>
   void FFT<1>::makePlans(RDField<1>& rField, RDFieldDft<1>& kField)
   {
      hipfftPlan1d(&fPlan_, rField.capacity(), HIPFFT_R2C, 1);
      hipfftPlan1d(&iPlan_, rField.capacity(), HIPFFT_C2R, 1);
   }

   template <>
   void FFT<2>::makePlans(RDField<2>& rField, RDFieldDft<2>& kField)
   {
      hipfftPlan2d(&fPlan_, rField.meshDimensions()[0], rField.meshDimensions()[1], HIPFFT_R2C);
      hipfftPlan2d(&iPlan_, rField.meshDimensions()[0], rField.meshDimensions()[1], HIPFFT_C2R);

   }

   template <>
   void FFT<3>::makePlans(RDField<3>& rField, RDFieldDft<3>& kField)
   {
      //std::cout<<"rfielddim2 "<<rField.meshDimensions()[2]<<std::endl;
      hipfftPlan3d(&fPlan_, rField.meshDimensions()[0], rField.meshDimensions()[1],
            rField.meshDimensions()[2], HIPFFT_R2C);
      hipfftPlan3d(&iPlan_, rField.meshDimensions()[0], rField.meshDimensions()[1],
            rField.meshDimensions()[2], HIPFFT_C2R);
   }

   // Explicit instantiation of relevant class instances
   template class FFT<1>;
   template class FFT<2>;
   template class FFT<3>;

}
}
